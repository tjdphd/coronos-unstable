#include "cls_fft_cuda_ext.hpp"

fft_cuda_ext::fft_cuda_ext() {

}

void fft_cuda_ext::cufftwInitialize( stack& run) {

  hipfftResult res_cufft;

  int n1;    run.stack_data.fetch("n1",   &n1   );
  int n2;    run.stack_data.fetch("n2",   &n2   );
  int n3;    run.stack_data.fetch("n3",   &n3   );

  int nr_in; run.stack_data.fetch("n1n2", &nr_in);
  int n1n2 ; run.stack_data.fetch("n1n2", &n1n2 );
  int n1n2c; run.stack_data.fetch("n1n2c", &n1n2c);
  int nc_out;

  nc_out    = n1 * (((int)(0.5*n2)) + 1);

  std::cout << "cufftwInitialize: nc_out = " << nc_out << std::endl;
  std::cout << "cufftwInitialize: n1n2c  = " << n1n2c  << std::endl;

  hipMalloc((void**)&cu_cplx_out,sizeof(hipfftDoubleComplex)*n1n2c);
  if ( hipGetLastError() != hipSuccess){ std::cout << "cufftwInitialize: unable to allocate cplx_out"     << std::endl;}

//  hipHostAlloc( (void**) &host_cplx_out, sizeof(ComplexVar)*n1n2c,hipHostMallocPortable );
//  hipHostAlloc( (void**) &host_real_in,  sizeof(RealVar)   *n1n2c,hipHostMallocPortable );

  host_cplx_out = (ComplexVar *)malloc(sizeof(ComplexVar)*n1n2c);
  host_real_in  = (RealVar    *)malloc(sizeof(RealVar)   *n1n2c);

  hipMalloc((void**)&cu_r_in,    sizeof(hipfftDoubleReal)   *n1n2 );
  if ( hipGetLastError() != hipSuccess){ std::cout << "cufftwInitialize: unable to allocate r_in"         << std::endl;}

  res_cufft = hipfftPlan2d(&cu_p_lay_for, n1, n2, HIPFFT_D2Z);
  if (res_cufft != HIPFFT_SUCCESS) { std::cout        << "cufftwInitialize: could not create plan cu_p_lay_for" << std::endl;}
  res_cufft = hipfftPlan2d(&cu_p_lay_rev, n1, n2, HIPFFT_Z2D);
  if (res_cufft != HIPFFT_SUCCESS) { std::cout        << "cufftwInitialize: could not create plan cu_p_lay_rev" << std::endl;}

}

void fft_cuda_ext::cufftwFinalize() {

  hipError_t res_err;
  hipfftResult res_cufft;

  res_cufft      = hipfftDestroy(cu_p_lay_for);
  if (res_cufft != HIPFFT_SUCCESS) {std::cout << "cufftwFinalize: could not destroy plan cu_p_lay_for" << std::endl;}
  res_cufft      = hipfftDestroy(cu_p_lay_rev);
  if (res_cufft != HIPFFT_SUCCESS) {std::cout << "cufftwFinalize: could not destroy plan cu_p_lay_rev" << std::endl;}

  res_err        = hipFree(cu_cplx_out);
  if (res_err   != hipSuccess) { std::cout  << "cufftwFinalize: unable to deallocate cplx_out"       << std::endl;}
  res_err        = hipFree(cu_r_in);
  if (res_err   != hipSuccess) { std::cout  << "cufftwFinalize: unable to deallocate r_in"           << std::endl;}

  free(host_cplx_out);
  free(host_real_in );

}

void fft_cuda_ext::cufftwForwardIC(RealArray& Rin, ComplexArray& Cout ) {

  int n1n2c; n1n2c = Cout.size();
  int n1n2;  n1n2  = Rin.size();

  RealVar scale    = ((RealVar) one)/((RealVar) (n1n2));

  for (unsigned k = 0 ; k < n1n2 ;  ++k) {host_real_in[k] =  Rin[k];                       }

  hipMemcpy(                 cu_r_in, host_real_in, ( sizeof(RealVar)*n1n2),hipMemcpyHostToDevice    );
  hipfftExecD2Z( cu_p_lay_for, cu_r_in, cu_cplx_out);
  hipMemcpy(host_cplx_out,            cu_cplx_out,  ( sizeof(ComplexVar)*n1n2c),hipMemcpyDeviceToHost);

  for (unsigned k = 0 ; k < n1n2c ; ++k) {Cout[k]         = scale * host_cplx_out[k]; }

  /* ~ should be able to access rt to allow for dealiasing (need to friend?) ~ */ 

}

void fft_cuda_ext::cufftwReverseIC(ComplexArray& Cin, RealArray& Rin )  {

}

fft_cuda_ext::~fft_cuda_ext() {

}
